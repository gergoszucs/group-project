#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "" // blockIdx gridDim etc.
#include "qmath.h"

__host__ __device__ void kernelCalcFiniteSvensonAt(const float *px, // the coordinates of the object panel control point (the target point).
	const float *py, // the coordinates of the object panel control point (the target point).
	const float *pz, // the coordinates of the object panel control point (the target point).
	const float *vs_x, // The vortex start point coordinates.
	const float *vs_y, // The vortex start point coordinates.
	const float *vs_z, // The vortex start point coordinates.
	const float *ve_x, // The vortex end point coordinates.
	const float *ve_y, // The vortex end point coordinates.
	const float *ve_z, // The vortex end point coordinates.
	float *dvx, // Returned variables; the cartesian components of the velocity induced at the object panel by the current finite vortex on the subject panel.
	float *dvy, // Returned variables; the cartesian components of the velocity induced at the object panel by the current finite vortex on the subject panel.
	float *dvz // Returned variables; the cartesian components of the velocity induced at the object panel by the current finite vortex on the subject panel.
)
{
	// This method returns a prediction of the velocity (in a ITPoint object) induced at the target point by this vortex.

	// Reference: 1. Sofia Werner MSc Report.
	//				 /Users/dominiquefleischmann/Documents/WorkingFolder/SWDevelopment/PCMAC/C++/Surfit Family Dev/Stuff/Documentation/sofiawernerMscreport.pdf

	/* This function computes the velocity at the point targetPoint induced by this
	* finite straight-line vortex segment using the closed form expression
	* due to Svenson and presented by Sofia Werner in Appendix A of her 2001 thesis.
	* Checked 20120708.
	*/

	// 20170203: Included denominator 4 PI.
	// Note that the induced velocity formulation used here excludes the multiplicative factor 1/(4 PI).
	float a, b, c, dx, dy, dz, term, sqrtABC, sqrtAterm, a1;

	// a is the square of the distance from the vortex start point to the object (target) point.
	// Since we are usually using a panel mid-point as a target point, the variable a will generally be greater than 0.
	a = (*px - *vs_x)*(*px - *vs_x)
		+ (*py - *vs_y)*(*py - *vs_y)
		+ (*pz - *vs_z)*(*pz - *vs_z);

	b = -2 * (
		(*ve_x - *vs_x)*(*px - *vs_x)
		+ (*ve_y - *vs_y)*(*py - *vs_y)
		+ (*ve_z - *vs_z)*(*pz - *vs_z)
		);

	// DOM: The length of the vortex squared.
	c = (*ve_x - *vs_x)*(*ve_x - *vs_x) + (*ve_y - *vs_y)*(*ve_y - *vs_y) + (*ve_z - *vs_z)*(*ve_z - *vs_z);

	dx = (*ve_z - *vs_z)*(*py - *vs_y)
		- (*ve_y - *vs_y)*(*pz - *vs_z);

	dy = (*ve_x - *vs_x)*(*pz - *vs_z)
		- (*ve_z - *vs_z)*(*px - *vs_x);

	dz = (*ve_y - *vs_y)*(*px - *vs_x)
		- (*ve_x - *vs_x)*(*py - *vs_y);

	term = 4 * a*c - b*b;

	if (term == 0.0) // Denominator.
	{
		*dvx = 0.0;
		*dvy = 0.0;
		*dvz = 0.0;

		return;
	}

	if (a <= 0.0) // Square of a denominator.
	{
		*dvx = 0.0;
		*dvy = 0.0;
		*dvz = 0.0;

		return;
	}

	if (a + b + c < 0.0) // Square of a denominator.
	{
		*dvx = 0.0;
		*dvy = 0.0;
		*dvz = 0.0;

		return;
	}

	sqrtABC = sqrtf(a + b + c);
	sqrtAterm = b / sqrtf(a);
	a1 = -2 * ((2 * c + b) / sqrtABC - sqrtAterm) / term;

	*dvx = dx * a1;
	*dvy = dy * a1;
	*dvz = dz * a1;

	// FIXME: This is a cludge to avoid NaN problems in the calling function. Not sure why this helps.
	if ((fabs(*dvx) > 1000.0) || (fabs(*dvy) > 1000.0) || (fabs(*dvz) > 1000.0))
	{
		*dvx = 0.0;
		*dvy = 0.0;
		*dvz = 0.0;

		return;
	}

	//if (custom_isnan(*dvx) || custom_isnan(*dvy) || custom_isnan(*dvz))
	//{
	//	*dvx = 0.0;
	//	*dvy = 0.0;
	//	*dvz = 0.0;

	//	cuPrintfDebug(2, "******** NAN\n", "", "", "", "", "", "");

	//	return;
	//}

	return;
} // End of kernelCalcFiniteSvensonAt.

__global__ void kernelInfluenceCoefficient(
	const float *px,
	const float *py,
	const float *pz,
	const float *nx,
	const float *ny,
	const float *nz,
	const float *vs_x,
	const float *vs_y,
	const float *vs_z,
	const float *ve_x,
	const float *ve_y,
	const float *ve_z,
	float *A,
	int noOfUnknownVortexStrengths,
	char rankineAlgorithmIndex,
	float RankineCoreRadius,
	int FrameNumber)
{
	// An instance of this kernel gets instanciated for each entry in the A matrix of influence coefficients.

	// =================================================================================
	// Input parameters:
	// px - the x-coordinate of the object panel control point.
	// py - the y-coordinate of the object panel control point.
	// pz - the z-coordinate of the object panel control point.
	// nx - the x-coordinate of the unit normal vector of the object panel.
	// ny - the y-coordinate of the unit normal vector of the object panel.
	// nz - the z-coordinate of the unit normal vector of the object panel.
	// vs_x - finite vortex start point.
	// vs_y - finite vortex start point.
	// vs_z - finite vortex start point.
	// ve_x - finite vortex end point.
	// ve_y - finite vortex end point.
	// ve_z - finite vortex end point.
	// noOfUnknownVortexStrengths - the row and column size of the matrix A.

	// Output parameter:
	// A - the entry in the A matrix corresponding to the index of this kernel instance.
	// =================================================================================

	// blockDim.x = number of threads in the block.
	// blockIdx.x = block index.
	// threadIdx.x = the thread index within the current block.

	int index = blockIdx.y  * gridDim.x  * blockDim.z * blockDim.y * blockDim.x
		+ blockIdx.x  * blockDim.z * blockDim.y * blockDim.x
		+ threadIdx.z * blockDim.y * blockDim.x
		+ threadIdx.y * blockDim.x
		+ threadIdx.x;

	if (index < noOfUnknownVortexStrengths*noOfUnknownVortexStrengths)  // DOM: Only do the computation if the index is valid.
																		// DOM: This conditional is needed in case the number of instances
																		//      of the kernel multiplied by noOfElementsInThisThread is greater than
																		//      the number of panels squared.
	{

		// DOM: Find the entry in the A matrix corresponding to the current value of index.
		// DOM: Note that in transient computations the b vector includes contributions from the wake.
		//      The b vector is computed in other code.
		int row = index / noOfUnknownVortexStrengths; // DOM: Object panel i. This is the row of the A matrix.
		int col = index % noOfUnknownVortexStrengths; // DOM: Subject panel j. This is the column of the A matrix.

		int maxNoOfVortices = 4; // This hard-coded number should be identical to the hard coded number on line 130 of ManageCalculationOfMatrixOfCoefficients in ITPhysics.cpp.

								 // DOM: Initialize the velocity induced at the row-th surface object panel due to the col-th surface subject panel.
		float Vx = 0.0;
		float Vy = 0.0;
		float Vz = 0.0;

		// DOM: Loop over all the finite vortices on the surface subject panel (4 segments on each panel).
		// DOM: Note that the contribution from the wake quadrilateral vortex loops is accounted for in the b vector.
		for (int n = 0; n < maxNoOfVortices; n++)
		{
			float dvx = 0.0;
			float dvy = 0.0;
			float dvz = 0.0;

			// 20160616.
			// Test for the zero vector.
			float epsilon = 0.001;
			if ((fabs(vs_x[col*maxNoOfVortices + n] - ve_x[col*maxNoOfVortices + n]) < epsilon)
				&& (fabs(vs_y[col*maxNoOfVortices + n] - ve_y[col*maxNoOfVortices + n]) < epsilon)
				&& (fabs(vs_z[col*maxNoOfVortices + n] - ve_z[col*maxNoOfVortices + n]) < epsilon))
			{
				// Points are (almost) colinear, so return with dv set to zero.
				return;
			}

			// Calculate the induced velocity at the colocation point of the object panel due to all the finite vortices on the current subject panel.
			// Use Biot-Savart vortex with Rankine viscous core.
			kernelCalcFiniteSvensonAt(
				&px[row],
				&py[row],
				&pz[row],
				&vs_x[col*maxNoOfVortices + n],
				&vs_y[col*maxNoOfVortices + n],
				&vs_z[col*maxNoOfVortices + n],
				&ve_x[col*maxNoOfVortices + n],
				&ve_y[col*maxNoOfVortices + n],
				&ve_z[col*maxNoOfVortices + n],
				&dvx,
				&dvy,
				&dvz);

			// DOM: Add the contribution to the induced velocity due
			// to the current vortex in the subject
			// panel vortex loop.
			Vx = Vx + dvx;
			Vy = Vy + dvy;
			Vz = Vz + dvz;
		}
		  // Now compute A[ index ].
		  // This is the dot-product of the
		  // induced velocity with the unit normal at 
		  // the target point (colocation point of the object panel).
		A[index] = (nx[row] * Vx + ny[row] * Vy + nz[row] * Vz);
	}
}

__global__ void kernelFunctionPredictVelocityAtPoint(
	const float *px,
	const float *py,
	const float *pz,
	const float *vs_x,
	const float *vs_y,
	const float *vs_z,
	const float *ve_x,
	const float *ve_y,
	const float *ve_z,
	float *p_vx,
	float *p_vy,
	float *p_vz,
	const float *vorticities,
	int noOfVorticesPerPanel,
	int noOfPanels,
	int noOfElementsInThisThread, // Stride.
	int noOfVelocityPredictions,
	int threadsPerTranche,
	int indexOfStartOfTranche,
	char rankineAlgorithmIndex)
{
	// Input parameters:
	// px - the x-coordinate of the object panel control point.
	// py - the y-coordinate of the object panel control point.
	// pz - the z-coordinate of the object panel control point.
	// vs_x - finite vortex start point.
	// vs_y - finite vortex start point.
	// vs_z - finite vortex start point.
	// ve_x - finite vortex end point.
	// ve_y - finite vortex end point.
	// ve_z - finite vortex end point.
	// vorticities - the values of vorticities on each panel.

	// Output parameters:
	// p_vx - The x-components of velocity at each object point.
	// p_vy - The y-components of velocity at each object point.
	// p_vz - The z-components of velocity at each object point.

	// maxNoOfFiniteVorticesPerPanel = maxNoOfVortices

	// blockDim.x = number of threads in the block.
	// blockIdx.x = block index.
	// threadIdx.x = the thread index within the current block.

	// Each instance of this kernel function is responsible for predicting the induced velocity at a single object point
	// as the result of the sum of all the vortex segments.

	// Be careful about evaluating the velocity near a vortex.

	// The index of the object point is threadIndex.

	int threadIndex = blockIdx.y  * gridDim.x  * blockDim.z * blockDim.y * blockDim.x
		+ blockIdx.x  * blockDim.z * blockDim.y * blockDim.x
		+ threadIdx.z * blockDim.y * blockDim.x
		+ threadIdx.y * blockDim.x
		+ threadIdx.x;

	// DOM: Initialize the free stream velocity vector.
	float Vx = 0.0f;
	float Vy = 0.0f;
	float Vz = 0.0f;

	// DOM: Only do the computation if we are in a valid thread.
	if (threadIndex < noOfVelocityPredictions)
	{
		// DOM: Only do the computation if the threadIndex is in this tranche.
		if ((threadIndex >= indexOfStartOfTranche) && (threadIndex < (indexOfStartOfTranche + threadsPerTranche)))
		{
			// DOM: Loop through the panels in the project.
			for (int panelIndex = 0; panelIndex < noOfPanels; panelIndex++) // Loop through all the subject panels.
			{
				// DOM: Loop through the vortices of the current panel.
				for (int vortexIndex = 0; vortexIndex < noOfVorticesPerPanel; vortexIndex++)
				{
					// Initialize the velocity induced by the current vortex at the object point.
					float dvx = 0.0;
					float dvy = 0.0;
					float dvz = 0.0;

					// DOM: Actually calculate the velocity induced at the object point due to the current vortex.
					// ROW_MAJOR_IDX2C(panelIndex, vortexIndex, maxNoOfVortices)
					// Avoid computing Svendsen induced velocity for object points close to the current vortex.
					float dsxSquared = (vs_x[panelIndex*noOfVorticesPerPanel + vortexIndex] - px[threadIndex]) * (vs_x[panelIndex*noOfVorticesPerPanel + vortexIndex] - px[threadIndex]);
					float dsySquared = (vs_y[panelIndex*noOfVorticesPerPanel + vortexIndex] - py[threadIndex]) * (vs_y[panelIndex*noOfVorticesPerPanel + vortexIndex] - py[threadIndex]);
					float dszSquared = (vs_z[panelIndex*noOfVorticesPerPanel + vortexIndex] - pz[threadIndex]) * (vs_z[panelIndex*noOfVorticesPerPanel + vortexIndex] - pz[threadIndex]);

					float dsSquared = dsxSquared + dsySquared + dszSquared;

					float dexSquared = (ve_x[panelIndex*noOfVorticesPerPanel + vortexIndex] - px[threadIndex]) * (ve_x[panelIndex*noOfVorticesPerPanel + vortexIndex] - px[threadIndex]);
					float deySquared = (ve_y[panelIndex*noOfVorticesPerPanel + vortexIndex] - py[threadIndex]) * (ve_y[panelIndex*noOfVorticesPerPanel + vortexIndex] - py[threadIndex]);
					float dezSquared = (ve_z[panelIndex*noOfVorticesPerPanel + vortexIndex] - pz[threadIndex]) * (ve_z[panelIndex*noOfVorticesPerPanel + vortexIndex] - pz[threadIndex]);

					float deSquared = dexSquared + deySquared + dezSquared;

					float eps = 0.0005;
					if ((dsSquared < eps) || (deSquared < eps))
					{
						// Object point is very close to one of the ends of the vortex, so leave induced velocity unchanged as zero. 
					}
					else
					{
						// Pass the addresses of array entries as parameters.
						kernelCalcFiniteSvensonAt(
							&px[threadIndex],
							&py[threadIndex],
							&pz[threadIndex],
							&vs_x[panelIndex*noOfVorticesPerPanel + vortexIndex],
							&vs_y[panelIndex*noOfVorticesPerPanel + vortexIndex],
							&vs_z[panelIndex*noOfVorticesPerPanel + vortexIndex],
							&ve_x[panelIndex*noOfVorticesPerPanel + vortexIndex],
							&ve_y[panelIndex*noOfVorticesPerPanel + vortexIndex],
							&ve_z[panelIndex*noOfVorticesPerPanel + vortexIndex],
							&dvx,
							&dvy,
							&dvz);
					}

					Vx = Vx + dvx*vorticities[panelIndex];
					Vy = Vy + dvy*vorticities[panelIndex];
					Vz = Vz + dvz*vorticities[panelIndex];
				}
			}

			p_vx[threadIndex] = Vx;
			p_vy[threadIndex] = Vy;
			p_vz[threadIndex] = Vz;
		}
	}
}