#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h> 
#include <cstdlib>
#include <iostream>
#include <time.h>

// Dom's includes.
#include "CudaPhysics.cuh"
#include "CudaKernels.cuh"

void ConstructMatrixOfInfluenceCoefficientsCuda(
	const float *h_cp_x,
	const float *h_cp_y,
	const float *h_cp_z,
	const float *h_n_x,
	const float *h_n_y,
	const float *h_n_z,
	const float *h_vs_x,
	const float *h_vs_y,
	const float *h_vs_z,
	const float *h_ve_x,
	const float *h_ve_y,
	const float *h_ve_z,
	float *h_A, // Output influence coefficient matrix.
	int noOfUnknownVortexStrengths, // This is basically the number of surface panels.
	float RankineCoreRadius,
	char rankineAlgorithmIndex,
	int FrameNumber)
{
	// DOM: Called from ManageCalculationOfMatrixOfCoefficients in ITPhysics.cpp.

	hipError_t err; // Error code to check return values for CUDA calls.

	// DOM: Calculate the sizes of the arrays passed in to this function.
	int totalNumberOfCudaComputations = noOfUnknownVortexStrengths*noOfUnknownVortexStrengths; // This is the number of elements in the A matrix, and accounts for the influence of each panel on each panel.

	size_t sizeRowFloat = noOfUnknownVortexStrengths * sizeof(float); // Memory required for a row of floats.
	size_t sizeMatrixFloat = totalNumberOfCudaComputations * sizeof(float); // Memory required for a matrix of floats.

	int maxNoOfVortices = 4;

	// ============================================================================
	// Allocate the GPU memory.
	// ============================================================================
	// Colocation point coordinates.
	float *d_cp_x = NULL;
	err = hipMalloc((void **)&d_cp_x, sizeRowFloat); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	float *d_cp_y = NULL;
	err = hipMalloc((void **)&d_cp_y, sizeRowFloat); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	float *d_cp_z = NULL;
	err = hipMalloc((void **)&d_cp_z, sizeRowFloat); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	// Panel Normals.
	float *d_n_x = NULL;
	err = hipMalloc((void **)&d_n_x, sizeRowFloat); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	float *d_n_y = NULL;
	err = hipMalloc((void **)&d_n_y, sizeRowFloat); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	float *d_n_z = NULL;
	err = hipMalloc((void **)&d_n_z, sizeRowFloat); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	// Vortex end point coordinates.
	float *d_vs_x = NULL;
	err = hipMalloc((void **)&d_vs_x, sizeRowFloat*maxNoOfVortices); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	float *d_vs_y = NULL;
	err = hipMalloc((void **)&d_vs_y, sizeRowFloat*maxNoOfVortices); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	float *d_vs_z = NULL;
	err = hipMalloc((void **)&d_vs_z, sizeRowFloat*maxNoOfVortices); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	float *d_ve_x = NULL;
	err = hipMalloc((void **)&d_ve_x, sizeRowFloat*maxNoOfVortices); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	float *d_ve_y = NULL;
	err = hipMalloc((void **)&d_ve_y, sizeRowFloat*maxNoOfVortices); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	float *d_ve_z = NULL;
	err = hipMalloc((void **)&d_ve_z, sizeRowFloat*maxNoOfVortices); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	// The memory for the square matrix of influence coefficient entries.
	float *d_A = NULL;
	err = hipMalloc((void **)&d_A, sizeMatrixFloat); if (err != hipSuccess) { exit(EXIT_FAILURE); }

	// ============================================================================
	// Copy host memory to device memory.
	// ============================================================================
	err = hipMemcpy(d_cp_x, h_cp_x, sizeRowFloat, hipMemcpyHostToDevice); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	err = hipMemcpy(d_cp_y, h_cp_y, sizeRowFloat, hipMemcpyHostToDevice); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	err = hipMemcpy(d_cp_z, h_cp_z, sizeRowFloat, hipMemcpyHostToDevice); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	err = hipMemcpy(d_n_x, h_n_x, sizeRowFloat, hipMemcpyHostToDevice); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	err = hipMemcpy(d_n_y, h_n_y, sizeRowFloat, hipMemcpyHostToDevice); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	err = hipMemcpy(d_n_z, h_n_z, sizeRowFloat, hipMemcpyHostToDevice); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	err = hipMemcpy(d_vs_x, h_vs_x, sizeRowFloat*maxNoOfVortices, hipMemcpyHostToDevice); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	err = hipMemcpy(d_vs_y, h_vs_y, sizeRowFloat*maxNoOfVortices, hipMemcpyHostToDevice); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	err = hipMemcpy(d_vs_z, h_vs_z, sizeRowFloat*maxNoOfVortices, hipMemcpyHostToDevice); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	err = hipMemcpy(d_ve_x, h_ve_x, sizeRowFloat*maxNoOfVortices, hipMemcpyHostToDevice); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	err = hipMemcpy(d_ve_y, h_ve_y, sizeRowFloat*maxNoOfVortices, hipMemcpyHostToDevice); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	err = hipMemcpy(d_ve_z, h_ve_z, sizeRowFloat*maxNoOfVortices, hipMemcpyHostToDevice); if (err != hipSuccess) { exit(EXIT_FAILURE); }

	err = hipMemcpy(d_A, h_A, sizeMatrixFloat, hipMemcpyHostToDevice); if (err != hipSuccess) { exit(EXIT_FAILURE); }


	// ============================================================================
	// Call the Kernel.
	// ============================================================================
	int noOfElementsInEachThread = 1;
	int totalNumberOfThreads = (totalNumberOfCudaComputations + noOfElementsInEachThread - 1) / noOfElementsInEachThread;

	int threadsPerBlock = 256; // 256; // When running out of resources, try reduce the threadsPerBlock.
	int totalNumberOfBlocks = (totalNumberOfThreads + threadsPerBlock - 1) / threadsPerBlock;
	int noOfBlocksX = 64;
	int noOfBlocksY = (totalNumberOfBlocks + noOfBlocksX - 1) / noOfBlocksX;

	dim3 grid(noOfBlocksX, noOfBlocksY, 1);
	dim3 block(threadsPerBlock, 1, 1);

	// Call noOfBlocksX*noOfBlocksY*threadsPerBlock instances of the kernel.
	kernelInfluenceCoefficient << <grid, block >> >(
		d_cp_x,
		d_cp_y,
		d_cp_z,
		d_n_x,
		d_n_y,
		d_n_z,
		d_vs_x,
		d_vs_y,
		d_vs_z,
		d_ve_x,
		d_ve_y,
		d_ve_z,
		d_A,
		noOfUnknownVortexStrengths,
		rankineAlgorithmIndex,
		RankineCoreRadius,
		FrameNumber);

	// Synchronize the CUDA kernels.
	hipDeviceSynchronize();

	// Deal with any errors.
	err = hipGetLastError();
	if (err != hipSuccess)
	{
		std::cout << "Failed to launch kernelInfluenceCoefficient kernel (error code " << hipGetErrorString(err) << ")" << std::endl;
		exit(EXIT_FAILURE);
	}

	// ====================================================================================================
	// Copy the coefficient vector back from the GPU device.
	// Copy the device result vector in device memory to the host result vector in host memory.
	// ====================================================================================================
	err = hipMemcpy(h_A, d_A, sizeMatrixFloat, hipMemcpyDeviceToHost);


	// ============================================================================
	// Free the GPU memory.
	// ============================================================================
	err = hipFree(d_cp_x); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	err = hipFree(d_cp_y); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	err = hipFree(d_cp_z); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	err = hipFree(d_n_x); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	err = hipFree(d_n_y); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	err = hipFree(d_n_z); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	err = hipFree(d_vs_x); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	err = hipFree(d_vs_y); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	err = hipFree(d_vs_z); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	err = hipFree(d_ve_x); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	err = hipFree(d_ve_y); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	err = hipFree(d_ve_z); if (err != hipSuccess) { exit(EXIT_FAILURE); }

	err = hipFree(d_A); if (err != hipSuccess) { exit(EXIT_FAILURE); }

}



void ComputeVelocitiesForBatchOfPointsCuda(
	const float *h_cp_x,
	const float *h_cp_y,
	const float *h_cp_z,
	const float *h_vs_x,
	const float *h_vs_y,
	const float *h_vs_z,
	const float *h_ve_x,
	const float *h_ve_y,
	const float *h_ve_z,
	float *h_cp_vx,
	float *h_cp_vy,
	float *h_cp_vz,
	const float *h_vorticities,
	int noOfVorticesPerPanel,
	int noOfSubjectPanels,
	int noOfVelocityPredictions,
	int rankineAlgorithmIndex)
{

	// DOM: Error code to check return values for CUDA calls.
	hipError_t err;

	// DOM: Calculate the sizes of the arrays passed in to this function.
	size_t sizeSubjectPanelsFloat = noOfSubjectPanels * sizeof(float); // Memory required for a row of floats.
	size_t sizeVelocityPredictionsFloat = noOfVelocityPredictions * sizeof(float); // Memory required for noOfVelocityPredictions floats.

	// ============================================================================
	// Allocate the GPU memory.
	// ============================================================================
	// Object point coordinates.
	float *d_cp_x = NULL;
	err = hipMalloc((void **)&d_cp_x, sizeVelocityPredictionsFloat); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	float *d_cp_y = NULL;
	err = hipMalloc((void **)&d_cp_y, sizeVelocityPredictionsFloat); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	float *d_cp_z = NULL;
	err = hipMalloc((void **)&d_cp_z, sizeVelocityPredictionsFloat); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	// Vortex end point coordinates.
	float *d_vs_x = NULL;
	err = hipMalloc((void **)&d_vs_x, sizeSubjectPanelsFloat*noOfVorticesPerPanel); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	float *d_vs_y = NULL;
	err = hipMalloc((void **)&d_vs_y, sizeSubjectPanelsFloat*noOfVorticesPerPanel); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	float *d_vs_z = NULL;
	err = hipMalloc((void **)&d_vs_z, sizeSubjectPanelsFloat*noOfVorticesPerPanel); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	float *d_ve_x = NULL;
	err = hipMalloc((void **)&d_ve_x, sizeSubjectPanelsFloat*noOfVorticesPerPanel); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	float *d_ve_y = NULL;
	err = hipMalloc((void **)&d_ve_y, sizeSubjectPanelsFloat*noOfVorticesPerPanel); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	float *d_ve_z = NULL;
	err = hipMalloc((void **)&d_ve_z, sizeSubjectPanelsFloat*noOfVorticesPerPanel); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	// The memory for the object point velocities.
	float *d_cp_vx = NULL;
	err = hipMalloc((void **)&d_cp_vx, sizeVelocityPredictionsFloat); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	float *d_cp_vy = NULL;
	err = hipMalloc((void **)&d_cp_vy, sizeVelocityPredictionsFloat); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	float *d_cp_vz = NULL;
	err = hipMalloc((void **)&d_cp_vz, sizeVelocityPredictionsFloat); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	// The memory for the panel vorticities
	float *d_vorticities = NULL;
	err = hipMalloc((void **)&d_vorticities, sizeSubjectPanelsFloat); if (err != hipSuccess) { exit(EXIT_FAILURE); }


	// ============================================================================
	// Copy host memory to device memory.
	// ============================================================================
	err = hipMemcpy(d_cp_x, h_cp_x, sizeVelocityPredictionsFloat, hipMemcpyHostToDevice); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	err = hipMemcpy(d_cp_y, h_cp_y, sizeVelocityPredictionsFloat, hipMemcpyHostToDevice); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	err = hipMemcpy(d_cp_z, h_cp_z, sizeVelocityPredictionsFloat, hipMemcpyHostToDevice); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	err = hipMemcpy(d_vs_x, h_vs_x, sizeSubjectPanelsFloat*noOfVorticesPerPanel, hipMemcpyHostToDevice); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	err = hipMemcpy(d_vs_y, h_vs_y, sizeSubjectPanelsFloat*noOfVorticesPerPanel, hipMemcpyHostToDevice); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	err = hipMemcpy(d_vs_z, h_vs_z, sizeSubjectPanelsFloat*noOfVorticesPerPanel, hipMemcpyHostToDevice); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	err = hipMemcpy(d_ve_x, h_ve_x, sizeSubjectPanelsFloat*noOfVorticesPerPanel, hipMemcpyHostToDevice); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	err = hipMemcpy(d_ve_y, h_ve_y, sizeSubjectPanelsFloat*noOfVorticesPerPanel, hipMemcpyHostToDevice); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	err = hipMemcpy(d_ve_z, h_ve_z, sizeSubjectPanelsFloat*noOfVorticesPerPanel, hipMemcpyHostToDevice); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	err = hipMemcpy(d_cp_vx, h_cp_vx, sizeVelocityPredictionsFloat, hipMemcpyHostToDevice); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	err = hipMemcpy(d_cp_vy, h_cp_vy, sizeVelocityPredictionsFloat, hipMemcpyHostToDevice); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	err = hipMemcpy(d_cp_vz, h_cp_vz, sizeVelocityPredictionsFloat, hipMemcpyHostToDevice); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	err = hipMemcpy(d_vorticities, h_vorticities, sizeSubjectPanelsFloat, hipMemcpyHostToDevice); if (err != hipSuccess) { exit(EXIT_FAILURE); }

	// =========================================================================================
	// Call the Kernel.
	// =========================================================================================
	int noOfElementsInEachThread = 1; // The number of object point array elements computed by each instance of the kernel function.
	int totalNumberOfThreads = (noOfVelocityPredictions + noOfElementsInEachThread - 1) / noOfElementsInEachThread;

	// TODO: Temporarily reduce threadsPerBlock to 1 to see if it helps with cuPrint.

	int threadsPerBlock = 1; // 256; // When running out of resources, try reduce the threadsPerBlock.
	int totalNumberOfBlocks = (totalNumberOfThreads + threadsPerBlock - 1) / threadsPerBlock;
	int noOfBlocksX = 64;
	int noOfBlocksY = (totalNumberOfBlocks + noOfBlocksX - 1) / noOfBlocksX;

	dim3 grid(noOfBlocksX, noOfBlocksY, 1);
	dim3 block(threadsPerBlock, 1, 1);

	// Initialize tranche variables for tranche execution.
	int threadsPerTranche = 6000;
	int noOfTranches = (noOfVelocityPredictions + threadsPerTranche - 1) / threadsPerTranche;

	for (int trancheIndex = 0; trancheIndex<noOfTranches; trancheIndex++)
	{
		clock_t time_end;
		time_end = clock() + 10 * CLOCKS_PER_SEC / 1000;
		while (clock() < time_end)
		{
		}

		// Sort out tranche start index.
		int indexOfStartOfTranche = trancheIndex*threadsPerTranche;

		// Call noOfBlocksX*noOfBlocksY*threadsPerBlock instances of the kernel.
		kernelFunctionPredictVelocityAtPoint << <grid, block >> >(
			d_cp_x,
			d_cp_y,
			d_cp_z,
			d_vs_x,
			d_vs_y,
			d_vs_z,
			d_ve_x,
			d_ve_y,
			d_ve_z,
			d_cp_vx,
			d_cp_vy,
			d_cp_vz,
			d_vorticities,
			noOfVorticesPerPanel,
			noOfSubjectPanels,
			noOfElementsInEachThread, // Usually set to 1.
			noOfVelocityPredictions,
			threadsPerTranche,
			indexOfStartOfTranche,
			rankineAlgorithmIndex);

		hipDeviceSynchronize();


		// Deal with any errors.
		err = hipGetLastError();
		if (err != hipSuccess)
		{
			std::cout << "Failed to launch kernelFunctionPredictVelocityAtPoint kernel (error code " << hipGetErrorString(err) << ")" << std::endl;
			exit(EXIT_FAILURE);
		}


	} // End of for tranches.

	err = hipMemcpy(h_cp_vx, d_cp_vx, sizeVelocityPredictionsFloat, hipMemcpyDeviceToHost);
	err = hipMemcpy(h_cp_vy, d_cp_vy, sizeVelocityPredictionsFloat, hipMemcpyDeviceToHost);
	err = hipMemcpy(h_cp_vz, d_cp_vz, sizeVelocityPredictionsFloat, hipMemcpyDeviceToHost);

	// Free the GPU memory.
	err = hipFree(d_cp_x); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	err = hipFree(d_cp_y); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	err = hipFree(d_cp_z); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	err = hipFree(d_vs_x); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	err = hipFree(d_vs_y); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	err = hipFree(d_vs_z); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	err = hipFree(d_ve_x); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	err = hipFree(d_ve_y); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	err = hipFree(d_ve_z); if (err != hipSuccess) { exit(EXIT_FAILURE); }

	err = hipFree(d_cp_vx); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	err = hipFree(d_cp_vy); if (err != hipSuccess) { exit(EXIT_FAILURE); }
	err = hipFree(d_cp_vz); if (err != hipSuccess) { exit(EXIT_FAILURE); }

	err = hipFree(d_vorticities); if (err != hipSuccess) { exit(EXIT_FAILURE); }

} // End of ComputeVelocitiesForBatchOfPointsCuda.