#include "hip/hip_runtime.h"
#include <iostream> 
#include <vector> 
#include <hip/hip_runtime.h> 
#include <sstream>

#pragma comment(lib, "cudart") 

using std::cerr;
using std::cout;
using std::endl;
using std::exception;
using std::vector;

static const int MaxSize = 96;

// CUDA kernel: cubes each array value 
__global__ void cubeKernel(float* result, float* data)
{
	int idx = threadIdx.x;
	float f = data[idx];
	result[idx] = f * f * f;
}

// Initializes data on the host 
void InitializeData(vector<float>& data)
{
	for (int i = 0; i < MaxSize; ++i)
	{
		data[i] = static_cast<float>(i);
	}
}

// Executes CUDA kernel 
void RunCubeKernel(vector<float>& data, vector<float>& result)
{
	const size_t size = MaxSize * sizeof(float);

	// TODO: test for error 
	float* d;
	float* r;
	hipError_t hr;

	hr = hipMalloc(reinterpret_cast<void**>(&d), size);            // Could return 46 if device is unavailable. 
	if (hr == cudaErrorDevicesUnavailable)
	{
		cerr << "Close all browsers and rerun" << endl;
		throw std::runtime_error("Close all browsers and rerun");
	}

	hr = hipMalloc(reinterpret_cast<void**>(&r), size);
	if (hr == cudaErrorDevicesUnavailable)
	{
		cerr << "Close all browsers and rerun" << endl;
		throw std::runtime_error("Close all browsers and rerun");
	}

	// Copy data to the device 
	hipMemcpy(d, &data[0], size, hipMemcpyHostToDevice);

	// Launch kernel: 1 block, 96 threads 
	// Important: Do not exceed number of threads returned by the device query, 1024 on my computer. 
	cubeKernel << <1, MaxSize >> > (r, d);

	// Copy back to the host 
	hipMemcpy(&result[0], r, size, hipMemcpyDeviceToHost);

	// Free device memory 
	hipFree(d);
	hipFree(r);
}

int GetDeviceCount()
{
	int devCount;
	hipGetDeviceCount(&devCount);
	return devCount;
}

std::string InitializeDevice(int argc, char *argv[], std::string *gpuString)
{
	std::string outputString;

	int devCount;
	hipGetDeviceCount(&devCount);

	for (int i = 0; i < devCount; ++i)
	{
		hipDeviceProp_t devProp;
		hipGetDeviceProperties(&devProp, i);

		gpuString->append(devProp.name);

		std::ostringstream s1;
		s1 << "Device index: " << i << std::endl;
		outputString.append(s1.str());

		std::ostringstream s2;
		s2 << "Major revision number: " << devProp.major << std::endl;
		outputString.append(s2.str());

		std::ostringstream s3;
		s3 << "Minor revision number: " << devProp.minor << std::endl;
		outputString.append(s3.str());

		std::ostringstream s4;
		s4 << "Name: " << devProp.name << std::endl;
		outputString.append(s4.str());

		std::ostringstream s5;
		s5 << "Total global memory: " << devProp.totalGlobalMem << std::endl;
		outputString.append(s5.str());

		std::ostringstream s6;
		s6 << "Total shared memory per block: " << devProp.sharedMemPerBlock << std::endl;
		outputString.append(s6.str());

		std::ostringstream s7;
		s7 << "Total registers per block: " << devProp.regsPerBlock << std::endl;
		outputString.append(s7.str());

		std::ostringstream s8;
		s8 << "Warp size: " << devProp.warpSize << " (should be 32)" << std::endl;
		outputString.append(s8.str());

		std::ostringstream s9;
		s9 << "Maximum memory pitch: " << devProp.memPitch << std::endl;
		outputString.append(s9.str());

		std::ostringstream s10;
		s10 << "Maximum threads per block: " << devProp.maxThreadsPerBlock << std::endl;
		outputString.append(s10.str());

		for (int j = 0; j < 3; ++j)
		{
			std::ostringstream s11;
			s11 << "Maximum dimension " << j << " of block: " << devProp.maxThreadsDim[j] << std::endl;
			outputString.append(s11.str());
		}

		for (int j = 0; j < 3; ++j)
		{
			std::ostringstream s12;
			s12 << "Maximum dimension " << j << " of grid: " << devProp.maxGridSize[j] << std::endl;
			outputString.append(s12.str());
		}

		std::ostringstream s13;
		s13 << "Clock rate: " << devProp.clockRate << std::endl;
		outputString.append(s13.str());

		std::ostringstream s14;
		s14 << "Total constant memory: " << devProp.totalConstMem << std::endl;
		outputString.append(s14.str());

		std::ostringstream s15;
		s15 << "Texture alignment: " << devProp.textureAlignment << std::endl;
		outputString.append(s15.str());

		std::ostringstream s16;
		s16 << "Concurrent copy and execution: " << (devProp.deviceOverlap ? "Yes" : "No") << std::endl;
		outputString.append(s16.str());

		std::ostringstream s17;
		s17 << "Number of multiprocessors: " << devProp.multiProcessorCount << std::endl;
		outputString.append(s17.str());

		std::ostringstream s18;
		s18 << "Kernel execution timeout: " << (devProp.kernelExecTimeoutEnabled ? "Yes" : "No") << std::endl << std::endl;
		outputString.append(s18.str());
	}

	return outputString;
}